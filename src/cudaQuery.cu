#include <iostream>
#include <memory>
#include <helper_string.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string>

#define SHOW_DETAIL 1

#define CUDA_ERROR_HANDLER(error_id,function_name) \
    if(error_id != hipSuccess) { \
        printf( #function_name " returned code:%d -> error_msg: %s\n",(int)error_id,hipGetErrorString(error_id) ); \ 
        printf("Result = FAIL\n"); \
    }
#define CUDA_PROP_PRINTF(property_name) \
    if(typeid(property_name)!=typeid(std::string)){ \
        std::string pro_value = std::to_string(property_name); \
        std::cout<<"cuda " #property_name "'s value:"<<pro_value<<std::endl;\
    } else { std::cout<<"cuda " #property_name "'s value:"<<property_name<<std::endl; }\
     

int main(int argc,char* argv[]) {
    // 开始
    printf("------ %s start ... ------ \n",argv[0]);
    // 查询设备数量
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);
    CUDA_ERROR_HANDLER(error_id,cudaGetdevice_count)
    if(device_count<=0) {
        printf("There are no available device(s) that support CUDA \n");
    }else {
        printf("Detected %d CUDA Capable device(s)\n", device_count);
    }
    int driver_version = 0;
    int runtime_version = 0;
    for(int dev=0;dev<device_count;++dev) {
        // 设置当前设备
        hipSetDevice(dev);
        // 查询对应基础版本
        hipDriverGetVersion(&driver_version);
        hipRuntimeGetVersion(&runtime_version);
        printf("  CUDA Driver Version/Runtime Version/NumDevs    %d.%d / %d.%d / %d\n", driver_version/1000, (driver_version%100)/10, runtime_version/1000, (runtime_version%100)/10,device_count);
        // 查询属性
        hipDeviceProp_t device_properties;
        hipGetDeviceProperties(&device_properties,dev);
        // 输出设备名称
        printf("  Device_properties.name :                       %s\n",device_properties.name);
#ifdef SHOW_DETAIL
        // 计算总的内存大小
        char msg[255];
        sprintf(msg,"  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)device_properties.totalGlobalMem/1048576.0f, (unsigned long long) device_properties.totalGlobalMem);
        printf(msg);
        // 内存刷新频率
        printf("  Memory Clock rate:                             %.0f Mhz\n", device_properties.memoryClockRate * 1e-3f);
        // 总线带宽
        printf("  Memory Bus Width:                              %d-bit\n",device_properties.memoryBusWidth);
        // 输出其支持的SM小代
        printf("  Support SM with:                               sm_%0x compute\n",((device_properties.major << 4) + device_properties.minor));
        //流式处理器数目/每个流式处理器中的core 数目/CUDA core的数目
        printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               device_properties.multiProcessorCount,
               _ConvertSMVer2Cores(device_properties.major, device_properties.minor),
               _ConvertSMVer2Cores(device_properties.major, device_properties.minor) * device_properties.multiProcessorCount);
        // GPU时钟频率
        printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", device_properties.clockRate * 1e-3f, device_properties.clockRate * 1e-6f);
        // 输出l2缓存大小
        if(device_properties.l2CacheSize) {
            printf("  L2 Cache Size:                                 %d bytes\n",device_properties.l2CacheSize);
        }
        // 查看纹理维度大小
        printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
               device_properties.maxTexture1D   , device_properties.maxTexture2D[0], device_properties.maxTexture2D[1],
               device_properties.maxTexture3D[0], device_properties.maxTexture3D[1], device_properties.maxTexture3D[2]);
        printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
               device_properties.maxTexture1DLayered[0], device_properties.maxTexture1DLayered[1]);
        printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
               device_properties.maxTexture2DLayered[0], device_properties.maxTexture2DLayered[1], device_properties.maxTexture2DLayered[2]);
        // 静态内存大小
        printf("  Total amount of constant memory:               %lu bytes\n", device_properties.totalConstMem);
        // 每个block的共享内存大小
        printf("  Total amount of shared memory per block:       %lu bytes\n", device_properties.sharedMemPerBlock);
        // 每个block的寄存器数量
        printf("  Total number of registers available per block: %d\n", device_properties.regsPerBlock);
        // GPU中最小调度单位warp的大小
        printf("  Warp size:                                     %d\n", device_properties.warpSize);
        // 每个流式处理器中的最大线线程数目
        printf("  Maximum number of threads per multiprocessor:  %d\n", device_properties.maxThreadsPerMultiProcessor);
        // 每个block中的线程数
        printf("  Maximum number of threads per block:           %d\n", device_properties.maxThreadsPerBlock);
        // 每个block中线程块的最大维度大小
        printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
               device_properties.maxThreadsDim[0],
               device_properties.maxThreadsDim[1],
               device_properties.maxThreadsDim[2]);
        // 每个网格的各个维度大小
        printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
               device_properties.maxGridSize[0],
               device_properties.maxGridSize[1],
               device_properties.maxGridSize[2]);
        // 最大内存步长
        printf("  Maximum memory pitch:                          %lu bytes\n", device_properties.memPitch);
        // 纹理对齐内存大小
        printf("  Texture alignment:                             %lu bytes\n", device_properties.textureAlignment);
        // 异步引擎计数
        printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (device_properties.deviceOverlap ? "Yes" : "No"), device_properties.asyncEngineCount);
        // 运行时超时限制
        printf("  Run time limit on kernels:                     %s\n", device_properties.kernelExecTimeoutEnabled ? "Yes" : "No");
        // 是否支持共享逻辑内存
        printf("  Integrated GPU sharing Host Memory:            %s\n", device_properties.integrated ? "Yes" : "No");
        // 是否支持共享内存映射
        printf("  Support host page-locked memory mapping:       %s\n", device_properties.canMapHostMemory ? "Yes" : "No");
        // 是否支持海浪处理
        printf("  Alignment requirement for Surfaces:            %s\n", device_properties.surfaceAlignment ? "Yes" : "No");
        // 是否支持ECC
        printf("  Device has ECC support:                        %s\n", device_properties.ECCEnabled ? "Enabled" : "Disabled");
        // 是否支持统一寻址
        printf("  Device supports Unified Addressing (UVA):      %s\n", device_properties.unifiedAddressing ? "Yes" : "No");
        // 是否支持协同内核启动
        printf("  Supports Cooperative Kernel Launch:            %s\n", device_properties.cooperativeLaunch ? "Yes" : "No");
        // 支持多设备协同内核启动
        printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n", device_properties.cooperativeMultiDeviceLaunch ? "Yes" : "No");
        // 对应的PCIE设备号和总线编号
        printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", device_properties.pciDomainID, device_properties.pciBusID, device_properties.pciDeviceID);
        // 支持的计算模型
        const char *sComputeMode[] =
        {
            "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
            "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
            "Prohibited (no host thread can use ::hipSetDevice() with this device)",
            "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
            "Unknown",
            NULL
        };
        printf("  Compute Mode:\n");
        printf("     < %s >\n", sComputeMode[device_properties.computeMode]);
        
    }
#else
        CUDA_PROP_PRINTF(device_properties.computeMode)
#endif
    // If there are 2 or more GPUs, query to determine whether RDMA is supported
    if (device_count >= 2)
    {
        hipDeviceProp_t prop[64];
        int gpuid[64]; // 找到对点传输的GPU
        int gpu_p2p_count = 0;

        for (int i=0; i < device_count; i++)
        {
            checkCudaErrors(hipGetDeviceProperties(&prop[i], i));

            // P2P先要确定其为费米或者开普勒架构
            if ((prop[i].major >= 2))
            {
                // This is an array of P2P capable GPUs
                gpuid[gpu_p2p_count++] = i;
            }
        }

        // Show all the combinations of support P2P GPUs
        int can_access_peer;
        // 支持数目大于2
        if (gpu_p2p_count >= 2)
        {
            for (int i = 0; i < gpu_p2p_count; i++)
            {
                for (int j = 0; j < gpu_p2p_count; j++)
                {
                    if (gpuid[i] == gpuid[j])
                    {
                        continue;
                    }
                    // 查询GPU之间是否支持直接通路
                    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, gpuid[i], gpuid[j]));
                        printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[i]].name, gpuid[i],
                           prop[gpuid[j]].name, gpuid[j] ,
                           can_access_peer ? "Yes" : "No");
                }
            }
        }
    }
    printf("------ %s end ... ------ \n\n",argv[0]);
    return 0;
}